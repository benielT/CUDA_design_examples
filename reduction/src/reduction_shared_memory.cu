#include "hip/hip_runtime.h"
#include <reduction_kernels.h>

__global__ void reduce_shared_kernel(float *arr, float *block_sum)
{
    extern __shared__ float shared_arr[];
    int local_id = threadIdx.x;
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

    shared_arr[local_id] = arr[thread_id];
  
    __syncthreads();

    for (unsigned int i = 1; i < blockDim.x; i *= 2) {
        if (local_id % (2 * i) == 0) {
            shared_arr[local_id] += shared_arr[local_id + i];
        }
        __syncthreads();
    }

    if (local_id == 0) {
        block_sum[blockIdx.x] = shared_arr[0];
    }
}

float reduce_shared(thrust::device_vector<float> dev_arr, thrust::device_vector<float> block_par_sum, int arr_size)
{
    int grid_size = (arr_size + MAX_BLOCK_SIZE - 1) / MAX_BLOCK_SIZE;
    // First reduction pass
    reduce_shared_kernel<<<grid_size, MAX_BLOCK_SIZE, MAX_BLOCK_SIZE * sizeof(float)>>>(
        dev_arr.data().get(), block_par_sum.data().get());
    hipDeviceSynchronize();

    // Second reduction pass
    while (grid_size > MAX_BLOCK_SIZE) {
        int new_grid_size = (grid_size + MAX_BLOCK_SIZE - 1) / MAX_BLOCK_SIZE;
        reduce_shared_kernel<<<new_grid_size, MAX_BLOCK_SIZE, MAX_BLOCK_SIZE * sizeof(float)>>>(
            block_par_sum.data().get(), block_par_sum.data().get());
        hipDeviceSynchronize();
        grid_size = new_grid_size;
    }
    // Final reduction pass
    reduce_shared_kernel<<<1, grid_size, grid_size * sizeof(float)>>>(
        block_par_sum.data().get(), dev_arr.data().get());
    hipDeviceSynchronize();

    float result = dev_arr[0];
    return result;
}

// float reduce_shared(thrust::device_vector<float> dev_arr, int arr_size)
// {
//     int grid_size = (arr_size + MAX_BLOCK_SIZE - 1) / MAX_BLOCK_SIZE;
//     thrust::device_vector<float> block_par_sum(grid_size);
//     reduce_shared_kernel<<<grid_size, MAX_BLOCK_SIZE, MAX_BLOCK_SIZE*sizeof(float)>>>(dev_arr.data().get(), block_par_sum.data().get());
//     reduce_shared_kernel<<<1, grid_size, grid_size*sizeof(float)>>>(block_par_sum.data().get(), dev_arr.data().get());
//     hipDeviceSynchronize();
//     float result = dev_arr[0];
//     return result;
// }