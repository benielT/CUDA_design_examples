#include "hip/hip_runtime.h"
#include <reduction_kernels.h>

__global__ void reduce_vanilla_kernel(float *arr, int m)
{
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    arr[thread_id] += arr[thread_id + m];
}

float reduce_vanilla(thrust::device_vector<float> dev_arr, int arr_size)
{
    for (int i = arr_size / 2; i > 0; i /= 2)
    {
        int threads = std::min(MAX_BLOCK_SIZE, i);
        int blocks = std::max(i/MAX_BLOCK_SIZE, 1);

        reduce_vanilla_kernel<<<blocks, threads>>>(dev_arr.data().get(), i);
    }
    hipDeviceSynchronize();
    float result = dev_arr[0];
    return result;
}